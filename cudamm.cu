#include "hip/hip_runtime.h"
#include <assert.h>
#include "matrix_utility.c"

#define TILE_WIDTH 16
#define BLOCK_SIZE 16
#define DEBUG_PRINT 0

/* Compile with nvcc -gencode=arch=compute_30,code=sm_30 cudamm.cu -o cudamm */

__global__ void MM_kernel(const float *mA, const float *mB, float *results, int Ah, int Aw, int Bw){

    __shared__ float Ads[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bds[TILE_WIDTH][TILE_WIDTH];
    
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    float result = 0.0f;

    for (int k = 0; k < ceil(Aw/(double)TILE_WIDTH); k++) {
        if (k * TILE_WIDTH + tx < Aw && Row < Ah) {
            Ads[ty][tx] = mA[Row * Aw + k * TILE_WIDTH + tx];
        } else {
            Ads[ty][tx] = 0.0f;
        }

        if (k * TILE_WIDTH + ty < Aw && Col < Bw) {
            Bds[ty][tx] = mB[(k * TILE_WIDTH + ty) * Bw + Col];
        } else {
            Bds[ty][tx] = 0.0f;
        }
        __syncthreads();

        for (int n = 0; n < TILE_WIDTH; ++n) { 
            result += Ads[ty][n] * Bds[n][tx];
        }
        __syncthreads();
    }

    if (Row < Ah && Col < Bw) {
        results[((by * blockDim.y + ty)*Bw)+(bx*blockDim.x)+tx] = result;
    }
}

void MM_dev(const float *mA, const float *mB, float *results, int Ah, int Aw, int Bw, double *gpuAllocTime, double *gpuCopyTime, double *gpuExecuteTime){
    //Allocate memory
    double alloc_start = getTime(); // Performance timer
    float *mA_dev,*mB_dev,*results_dev;
    
    assert(hipMalloc((void**) &mA_dev,sizeof(float)*(Ah*Aw)) == hipSuccess);
    assert(hipMalloc((void**) &mB_dev,sizeof(float)*(Aw*Bw)) == hipSuccess);
    assert(hipMalloc((void**) &results_dev,sizeof(float)*(Ah*Bw)) == hipSuccess);
    
    double alloc_stop       = getTime(); // Performance timer
    double copy_in_start    = getTime(); // Performance timer
    
    //copy the input matrices to the device
    assert(hipMemcpy(mA_dev,mA,sizeof(float)*(Ah*Aw),hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(mB_dev,mB,sizeof(float)*(Aw*Bw),hipMemcpyHostToDevice) == hipSuccess);
    
    double copy_in_stop = getTime(); // Performance timer
    double kernel_start = getTime(); // Performance timer
    
    //invoke the kernel
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE,1);
    dim3 dimGrid(ceil(Bw/(double)dimBlock.x),ceil(Ah/(double)dimBlock.y),1);
    MM_kernel<<<dimGrid,dimBlock>>>(mA_dev,mB_dev,results_dev,Ah,Aw,Bw);

    double kernel_stop      = getTime(); // Performance timer
    double copy_out_start   = getTime(); // Performance timer
    
    //retrieve results
    assert(hipMemcpy(results,results_dev,sizeof(float)*(Ah*Bw),hipMemcpyDeviceToHost) == hipSuccess);

    double copy_out_stop = getTime(); // Performance timer
    
    *gpuAllocTime    = alloc_stop - alloc_start;
    *gpuCopyTime     = copy_in_stop - copy_in_start + copy_out_stop - copy_out_start;
    *gpuExecuteTime  = kernel_stop - kernel_start;
    
    //free device memory
    hipFree(mA_dev);
    hipFree(mB_dev);
    hipFree(results_dev);    
}

int main(int argc, char *argv[]){

    if(argc < 5){
        printf("You forgot to enter the matrix dimensions!\n");
        return 0;
    }
    if(argc > 5){
        printf("Too many arguments!\n");
        return 0;
    }
    
    srand(time(NULL));
    // Timer result variables
    double gpuAllocateTime, gpuCopyTime, gpuExecuteTime, fillMatrixTime, cpuAllocateTime, totalDeviceTime, totalTime;
    
    double total_start = getTime();
    
    // store command line arguments as the matrix dimensions
    int Ah = atoi(argv[1]);
    int Aw = atoi(argv[2]);
    int Bh = atoi(argv[3]);
    int Bw = atoi(argv[4]);
    if(Ah % BLOCK_SIZE || Aw % BLOCK_SIZE || Bh % BLOCK_SIZE || Bw % BLOCK_SIZE) {
        printf("Inputs must be multiples of %d.\n", BLOCK_SIZE);
        return 0;
    }
    
    //if the given dimensions are not multiply-able give error and exit
    if(Aw != Bh){
        printf("Cannot compute the matrix multiplication given the matrix dimensions\n");
        return 0;
    }
    
    double cpu_allocate_start = getTime();
    
    // setup matrices
    float *A, *B, *results, *check;
    char *id;    

    A = (float*)malloc(sizeof(float)*Ah*Aw); assert(A != 0);
    B = (float*)malloc(sizeof(float)*Bh*Bw); assert(B != 0);
    results = (float*)malloc(sizeof(float)*Ah*Bw); assert(results != 0);
    check = (float*)malloc(sizeof(float)*Ah*Bw); assert(check != 0);
    
    double cpu_allocate_stop    = getTime();
    cpuAllocateTime             = cpu_allocate_stop - cpu_allocate_start;
    double fill_start           = getTime();
    
    // fill the matrices with random values
    fillMatrix(Ah, Aw, A);
    fillMatrix(Bh, Bw, B);
    
    double fill_stop = getTime();
    fillMatrixTime   = fill_stop - fill_start;
    
    // display the matrices
    if(DEBUG_PRINT) {
        id = "A";
        printMatrix(Ah,Aw,id,A);
        id = "B";
        printMatrix(Bh,Bw,id,B);
    }
    
    // multiply on the device
    double device_start = getTime();
    MM_dev(A,B,results,Ah,Aw,Bw,&gpuAllocateTime,&gpuCopyTime,&gpuExecuteTime);
    double device_stop = getTime();
    totalDeviceTime = device_stop - device_start;
    
    // multiply on the host
    //double host_start = getTime();
    //cpuMatrixMultiply(A,B,check,Ah,Aw,Bw,&cpuExecuteTime);
    //double host_stop = getTime();
    //totalHostTime = host_stop - host_start;
    
    // display the results
    if(DEBUG_PRINT) {
        id = "results";
        printMatrix(Ah,Bw,id,results);
        id = "check";
        printMatrix(Ah,Bw,id,check);
    }
    
    // compare the matrices and display a message if they are not identical
    //double compare_start = getTime();
    ///int isSame = compareMatrices(results, check, Bw, Ah);
    //double compare_stop = getTime();
    //compareTime = compare_stop - compare_start;
    
    free(A);
    free(B);
    free(results);
    free(check);
    
    double total_stop   = getTime();
    totalTime           = total_stop - total_start;
    
    if(1) {
        printf("%d\t%d\t%d\t%d\t%d\t%d\t%f\t%f\t%f\t%f\t%f\t%f\t%f\n",Ah,Aw,Bh,Bw,Ah,Bw,gpuAllocateTime, gpuCopyTime, gpuExecuteTime, fillMatrixTime, cpuAllocateTime, totalDeviceTime, totalTime);
    } else {
        printf("The matrices do not match!\n");
    }
    
    return 0;
}
